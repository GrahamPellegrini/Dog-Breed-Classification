
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_cuda() {
    printf("Hello from CUDA thread %d\n", threadIdx.x);
}

int main() {
    hello_cuda<<<1, 10>>>();
    hipDeviceSynchronize();
    return 0;
}
